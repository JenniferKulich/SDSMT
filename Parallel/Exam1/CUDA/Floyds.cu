#include "hip/hip_runtime.h"
 /*****************************************************************************
  * Floyds.cu
  * Author: Jennifer Kulich
  * Class: CSC 410
  * Due: Oct 19, 2020
  * 
  * This will implement Floyd's algorithm using CUDA for parallelization. Description
  * of the program and what it does is in the pdf 
  *
  *
  ****************************************************************************/
#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <omp.h>
#include <stdlib.h>
#include <math.h>
#include <limits.h>

int THREAD_COUNT = 512;
int NUMBER_VERTICES = 6;
#define INF INT_MAX
int VERTICES = 6;

void printSolution(int graph[][VERTICES]);
void InnerLoop(int** graph, int k);

double floydsAlgorithm(int** graph)
{
	int dist[VERTICES][VERTICES], i, j, k;

	for(i = 0; i < VERTICES; i++)
	{
		for(j = 0; j < VERTICES; j++)
		{
			dist[i][j] = graph[i][j];
		}
	}

	//this is where I would copy the graph to the GPU

	for(k = 0; k < VERTICES; k++)
	{
		InnerLoop<<<dim3(x,y,z),dim3(a,b,c)>>>(dist, k);
		hipDeviceSynchronize();
	}

	//This is where I would copy it back to the host

	printSolution(dist);
	return totalTime;
}

__global__ void InnerLoop(int **graph, int k)
{	
	//in here, I would find an index using the blockID, blockDimension, and threadIDx
	int t;

	for(int i = 0; i < VERTICES, i++)
	{
		for(in j = 0; j < VERTICES; j++)
		{
			t = graph[i][k] + graph[k][j];
			graph[i][j] = t * (t<graph[i][j]) + graph[i][j] * (t >= graph[i][j]);i 			
		}
}

void printSolution(int graph[][VERTICES])
{
	for(int i = 0; i < VERTICES; i++)
	{
		for(int j = 0; j < VERTICES; j++)
		{
			if(graph[i][j] == INF)
				printf("%7s", "INF");
			else
				printf("%7d", graph[i][j]);
		}
		printf("\n");
	}
}


int main(int argc, char **argv)
{
	
	if(argc != 2)
	{
		printf("Wrong number of arguments\n");
		return -1;
	}

	VERTICES = strtol(argv[1], NULL, 10);

	int **graph;
	int i,j;
	time_t t;

	graph = malloc(VERTICES*sizeof(int *));
	if(graph == NULL)
		return -1;
	for(i = 0; i < VERTICES; i++)
	{
		graph[i] = malloc(VERTICES*sizeof(int));
		if(graph[i] == NULL)
			return -1;
	}

	srand((unsigned) time(&t));

	//randonly assign graph
//	for(i = 0; i < VERTICES; i++)
//	{
//		for(j = 0; j < VERTICES; j++)
//		{
//			if(i == j)
//				graph[i][j] = 0;
//			else
//			{
//				if(rand() % 4 == 0)
//					graph[i][j] = (rand() % 20) + 1;
//				else
//					graph[i][j] = INF;
//			}
//
//		}
//	}
//
//	for(int i = 0; i < VERTICES; i++)
//	{
//		for(j = 0; j < VERTICES; j++)
//		{
//			if(graph[i][j] == INF)
//                                printf("%7s", "INF");
//                         else
//                                 printf("%7d", graph[i][j]);
//                 }
//                 printf("\n");

//	}

	printf("\n\n");
	graph[0][0] = 0;
	graph[0][1] = 2;
	graph[0][2] = 4;
	graph[0][3] = INF;
	graph[0][4] = INF;
	graph[0][5] = INF;

	graph[1][0] = INF; 
	graph[1][1] =0;
        graph[1][2] = 7;
        graph[1][3] = 1;
        graph[1][4] = INF;
        graph[1][5] = 8;

	graph[2][0] = INF;
        graph[2][1] = INF;
        graph[2][2] = 0;
        graph[2][3] = 4;
        graph[2][4] = INF;
        graph[2][5] = INF;

	graph[3][0] = INF;
	graph[3][1] = INF;
	graph[3][2] = INF;
	graph[3][3] = 0;
	graph[3][4] = 3;
	graph[3][5] = INF;

	graph[4][0] = INF;
	graph[4][1] = INF;
	graph[4][2] = 2;
	graph[4][3] = INF;
	graph[4][4] = 0;
	graph[4][5] = 3;

	graph[5][0] = INF;
	graph[5][1] = 5;
	graph[5][2] = INF;
	graph[5][3] = 2;
	graph[5][4] = 4;
	graph[5][5] = 0;
	

	return 0;
}
